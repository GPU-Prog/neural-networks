#include "hip/hip_runtime.h"
#include "matrix.h"
#include <stdlib.h>
#include <string.h>
#include "error.h"
#include <hipblas.h>


#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
#define TILE_SIZE 16

/**
 * Libère la mémoire GPU associée à une matrice. 
 * Désalloue d'abord les données, puis la structure.
 */
void destroy_matrix(matrix_t *m)
{
    //printf("free %p %p\n", m, m->m);

    CHECK_ERROR(hipFree(m->m));
    CHECK_ERROR(hipFree(m));
}

/**
 * Alloue une matrice en mémoire GPU et initialise ses valeurs à zéro.
 */
matrix_t * alloc_matrix(unsigned rows, unsigned columns)
{
    matrix_t * res;
    CHECK_ERROR(hipMallocManaged((void**)&res, sizeof(matrix_t)));
    //matrix_t * res = (matrix_t*) malloc( sizeof(matrix_t) );

    //res->m = (double *) calloc(columns * rows, sizeof(double));
    CHECK_ERROR(hipMallocManaged((void**)&res->m, rows*columns*sizeof(double)));
    CHECK_ERROR(hipMemset(res->m, 0, rows*columns*sizeof(double)));

    res->columns = columns;
    res->rows = rows;
    return res;
}

/**
 * Affiche une matrice avec une option d'affichage réduit.
 */
void print_matrix(matrix_t *m, bool is_short){
    unsigned lim_rows = 0;
    unsigned lim_col = 0;

    if (is_short)
    {
        lim_rows = MIN(m->rows, 4);
        lim_col = MIN(m->columns, 10);
    }
    else
    {
        lim_rows = m->rows;
        lim_col = m->columns;
    }

    for (int row = 0; row < lim_rows; row ++)
    {
        for (int col = 0; col < lim_col; col ++)
        {
            printf("%.2lf ", m->m[col + row * m->columns]);
        }
        if (is_short && lim_col != m->columns) printf("...");
        printf("\n");
    }
    if (is_short && lim_rows != m->rows) printf("...\n");
}

/**
 * CPU
 * Calcule le produit de Hadamard de deux matrices et stocke le résultat.
 */
void hadamard_product(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)   &&
             (m1->columns == res->columns)  &&
             (m1->rows == m2->rows)         &&
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
            res->m[idx] = m1->m[idx] * m2->m[idx];
    }
}

/**
 * GPU en utilisant la parallélisation CUDA
 * Calcule le produit de Hadamard de deux matrices et stocke le résultat.
 */
__global__
void hadamard_product_GPU(matrix_t *m1, matrix_t *m2, matrix_t *res) {

    assert ( (m1->columns == m2->columns)   &&
             (m1->columns == res->columns)  &&
             (m1->rows == m2->rows)         &&
             (m1->rows == res->rows));

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m1->rows && col < m2->columns) {
        res->m[row*res->columns+col] = m1->m[row*m1->columns+col] * m2->m[row*m2->columns+col];
    } 
}

/**
 * CPU
 * Calcule la somme élément par élément de deux matrices.
 */
void matrix_sum(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    { 
        res->m[idx] = m1->m[idx] + m2->m[idx];
    }
}

/**
 * GPU en utilisant la parallélisation CUDA
 * Calcule la somme élément par élément de deux matrices.
 */
__global__
void matrix_sum_GPU(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row*m1->columns+col;

    if (row < m1->rows && col < m1->columns) {
        res->m[idx] = m1->m[idx] + m2->m[idx];
    } 
}

/**
 * CPU
 * Calcule la différence élément par élément de deux matrices.
 */
void matrix_minus(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));
             
    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
        res->m[idx] = m1->m[idx] - m2->m[idx];
    }
}

/**
 * GPU en utilisant la parallélisation CUDA
 * Calcule la différence élément par élément de deux matrices.
 */
__global__
void matrix_minus_GPU(matrix_t *m1, matrix_t *m2, matrix_t *res) {

    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row*m1->columns+col;

    if (row < m1->rows && col < m1->columns) {
        res->m[idx] = m1->m[idx] - m2->m[idx];
    } 
}

/**
 * CPU
 * Effectue le produit matriciel classique entre deux matrices.
 */
void matrix_dot(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->rows)  &&
             (m1->rows == res->rows)    &&
             (m2->columns == res->columns));

    for (int row = 0; row < m1->rows; row ++)
    {
        for (int col = 0; col < m2->columns; col ++)
        {
            int idx = col + row * m2->columns;
            double var = 0.0;

            for (int ii = 0; ii < m1->columns; ii++)
            {
                var += m1->m[ii + row * m1->columns] * m2->m[col + ii * m2->columns];
            }

            res->m[idx] = var;
        }
    }
}

/**
 * GPU en utilisant la parallélisation CUDA
 * Effectue le produit matriciel classique entre deux matrices.
 */
__global__
void matrix_dot_GPU(matrix_t *m1, matrix_t *m2, matrix_t *res) {

    assert ( (m1->columns == m2->rows)  &&
    (m1->rows == res->rows)    &&
    (m2->columns == res->columns));

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m1->rows && col < m2->columns)
    {
        float sum = 0;
        for (int i = 0; i < m1->columns; i++)
        {
            sum += m1->m[row*m1->columns+i] * m2->m[i*m2->columns+col];
        }
        res->m[row*m2->columns+col] = sum;
    } 
}

/**
 * GPU en utilisant la multiplication en tuiles
 * Effectue le produit matriciel classique entre deux matrices.
 */
__global__
void matrix_dot_GPU_shared(matrix_t *m1, matrix_t *m2, matrix_t *res) {
    assert((m1->columns == m2->rows) &&
           (m1->rows == res->rows) &&
           (m2->columns == res->columns));

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    float sum = 0.0f;
    int tileCount = (m1->columns + TILE_SIZE - 1) / TILE_SIZE;

    for (int t = 0; t < tileCount; t++) {
        if (row < m1->rows && t * TILE_SIZE + threadIdx.x < m1->columns)
            tileA[threadIdx.y][threadIdx.x] = m1->m[row * m1->columns + t * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < m2->columns && t * TILE_SIZE + threadIdx.y < m2->rows)
            tileB[threadIdx.y][threadIdx.x] = m2->m[(t * TILE_SIZE + threadIdx.y) * m2->columns + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < m1->rows && col < m2->columns) {
        res->m[row * m2->columns + col] = sum;
    }
}

/**
 * Effectue le produit matriciel en utilisant la bibliothèque cuBLAS.
 */
void matrix_dot_cublas(matrix_t* m1, matrix_t* m2, matrix_t* res) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const double alpha = 1.0;
    const double beta = 0.0;

    hipblasDgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                m1->rows, m2->columns, m1->columns,
                &alpha,
                m1->m, m1->rows,
                m2->m, m2->rows,
                &beta,
                res->m, res->rows);

    hipblasDestroy(handle);
}

/**
 * Applique une fonction scalaire à chaque élément d'une matrice.
 */
void matrix_function(matrix_t *m1, double (*f)(double), matrix_t *res)
{
    assert ( (m1->columns == res->columns) &&             
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
        res->m[idx] = f(m1->m[idx]);
    }
}

/**
 * CPU
 * Calcule la transposée d'une matrice.
 */
void matrix_transpose(matrix_t *m1, matrix_t *res)
{
    assert ( (m1->columns == res->rows) &&             
             (m1->rows == res->columns));
    
    for (int row = 0; row < m1->rows; row++)
    {
        for (int col = 0; col < m1->columns; col ++)
        {
            res->m[row + col * m1->rows] = m1->m[col + row * m1->columns];
        }
    }
}

/**
 * GPU en utilisant la parallélisation CUDA
 * Calcule la transposée d'une matrice.
 */
__global__
void matrix_transpose_GPU(matrix_t *m1, matrix_t *res) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m1->rows && col < m1->columns)
    {
        res->m[col*m1->rows+row] = m1->m[row*m1->columns+col];
    } 
}

/**
 * CPU
 * Multiplie chaque élément d'une matrice par un scalaire.
 */
void matrix_scalar(matrix_t *m1, double s, matrix_t *res)
{
    assert ( (m1->rows == res->rows) &&             
             (m1->columns == res->columns));

    for (int idx = 0; idx < m1->columns*m1->rows; idx ++)
    {
        res->m[idx] = m1->m[idx] * s;
    }
}

/**
 * GPU en utilisant la parallélisation CUDA
 * Multiplie chaque élément d'une matrice par un scalaire.
 */
__global__
void matrix_scalar_GPU(matrix_t *m1, double s, matrix_t *res) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row*res->columns+col;

    if (row < m1->rows && col < m1->columns) {
        res->m[idx] = m1->m[idx] * s;
    } 

}

/**
 * Copie les éléments d'une matrice source vers une matrice destination.
 */
void matrix_memcpy(matrix_t *dest, const matrix_t *src)
{
    assert ( (dest->rows == src->rows)      &&             
             (dest->columns == src->columns));

    memcpy(dest->m, src->m, src->columns * src->rows * sizeof(double));     
}